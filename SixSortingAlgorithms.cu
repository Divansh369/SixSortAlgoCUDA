#include <iostream>
#include <chrono>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>
#include <iomanip>

using namespace std;

// Function to generate a random array
vector<int> generate_random_array(int size) {
    vector<int> arr(size);
    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<int> dis(-10000 , 10000);
    for (int i = 0; i < size; ++i) {
        arr[i] = dis(gen);
    }
    return arr;
}

// Quick Sort Serial
void quick_sort_serial(vector<int>& arr, int low, int high) {
    if (low < high) {
        int pivot = arr[high];
        int i = low - 1;
        for (int j = low; j <= high - 1; j++) {
            if (arr[j] < pivot) {
                i++;
                swap(arr[i], arr[j]);
            }
        }
        swap(arr[i + 1], arr[high]);
        int pi = i + 1;

        quick_sort_serial(arr, low, pi - 1);
        quick_sort_serial(arr, pi + 1, high);
    }
}

// Merge Sort Serial
void merge(vector<int>& arr, int l, int m, int r) {
    int n1 = m - l + 1;
    int n2 = r - m;

    vector<int> L(n1), R(n2);
    for (int i = 0; i < n1; i++)
        L[i] = arr[l + i];
    for (int j = 0; j < n2; j++)
        R[j] = arr[m + 1 + j];

    int i = 0;
    int j = 0;
    int k = l;
    while (i < n1 && j < n2) {
        if (L[i] <= R[j]) {
            arr[k] = L[i];
            i++;
        }
        else {
            arr[k] = R[j];
            j++;
        }
        k++;
    }

    while (i < n1) {
        arr[k] = L[i];
        i++;
        k++;
    }

    while (j < n2) {
        arr[k] = R[j];
        j++;
        k++;
    }
}

void merge_sort_serial(vector<int>& arr, int l, int r) {
    if (l >= r) {
        return;
    }
    int m = l + (r - l) / 2;
    merge_sort_serial(arr, l, m);
    merge_sort_serial(arr, m + 1, r);
    merge(arr, l, m, r);
}

// Heap Sort Serial
void heapify(vector<int>& arr, int n, int i) {
    int largest = i;
    int l = 2 * i + 1;
    int r = 2 * i + 2;

    if (l < n && arr[l] > arr[largest])
        largest = l;

    if (r < n && arr[r] > arr[largest])
        largest = r;

    if (largest != i) {
        swap(arr[i], arr[largest]);
        heapify(arr, n, largest);
    }
}

void heap_sort_serial(vector<int>& arr) {
    int n = arr.size();

    for (int i = n / 2 - 1; i >= 0; i--)
        heapify(arr, n, i);

    for (int i = n - 1; i > 0; i--) {
        swap(arr[0], arr[i]);
        heapify(arr, i, 0);
    }
}

// Bubble Sort Serial
void bubble_sort_serial(vector<int>& arr) {
    int n = arr.size();
    for (int i = 0; i < n - 1; i++) {
        for (int j = 0; j < n - i - 1; j++) {
            if (arr[j] > arr[j + 1]) {
                swap(arr[j], arr[j + 1]);
            }
        }
    }
}

// Selection Sort Serial
void selection_sort_serial(vector<int>& arr) {
    int n = arr.size();
    for (int i = 0; i < n - 1; i++) {
        int min_idx = i;
        for (int j = i + 1; j < n; j++) {
            if (arr[j] < arr[min_idx]) {
                min_idx = j;
            }
        }
        swap(arr[i], arr[min_idx]);
    }
}

// Insertion Sort Serial
void insertion_sort_serial(vector<int>& arr) {
    int n = arr.size();
    for (int i = 1; i < n; i++) {
        int key = arr[i];
        int j = i - 1;
        while (j >= 0 && arr[j] > key) {
            arr[j + 1] = arr[j];
            j = j - 1;
        }
        arr[j + 1] = key;
    }
}

// Bubble Sort Parallel (CUDA)
__global__ void bubble_sort_kernel(int* arr, int n) {
    int idx = threadIdx.x;
    if (idx < n) {
        for (int i = 0; i < n - 1; i++) {
            if (idx < n - i - 1 && arr[idx] > arr[idx + 1]) {
                int temp = arr[idx];
                arr[idx] = arr[idx + 1];
                arr[idx + 1] = temp;
            }
        }
    }
}

// Selection Sort Parallel (CUDA)
__global__ void selection_sort_kernel(int* arr, int n) {
    int idx = threadIdx.x;
    if (idx < n - 1) {
        int min_idx = idx;
        for (int j = idx + 1; j < n; j++) {
            if (arr[j] < arr[min_idx]) {
                min_idx = j;
            }
        }
        int temp = arr[idx];
        arr[idx] = arr[min_idx];
        arr[min_idx] = temp;
    }
}

// Insertion Sort Parallel (CUDA)
__global__ void insertion_sort_kernel(int* arr, int n) {
    int idx = threadIdx.x;
    if (idx > 0 && idx < n) {
        int key = arr[idx];
        int j = idx - 1;
        while (j >= 0 && arr[j] > key) {
            arr[j + 1] = arr[j];
            j = j - 1;
        }
        arr[j + 1] = key;
    }
}

// CUDA kernel for parallel Quick Sort
__device__ void quick_sort_kernel(int* arr, int low, int high) {
    if (low < high) {
        int pivot = arr[high];
        int i = low - 1;
        for (int j = low; j <= high - 1; j++) {
            if (arr[j] < pivot) {
                i++;
                int temp = arr[i];
                arr[i] = arr[j];
                arr[j] = temp;
            }
        }
        int temp = arr[i + 1];
        arr[i + 1] = arr[high];
        arr[high] = temp;

        int pi = i + 1;

        quick_sort_kernel(arr, low, pi - 1);
        quick_sort_kernel(arr, pi + 1, high);
    }
}

__global__ void parallel_quick_sort_kernel(int* arr, int size) {
    quick_sort_kernel(arr, 0, size - 1);
}

// CUDA kernel for parallel Merge Sort
__device__ void merge_kernel(int* arr, int l, int m, int r) {
    int n1 = m - l + 1;
    int n2 = r - m;

    int* L = new int[n1];
    int* R = new int[n2];

    for (int i = 0; i < n1; i++)
        L[i] = arr[l + i];
    for (int j = 0; j < n2; j++)
        R[j] = arr[m + 1 + j];

    int i = 0;
    int j = 0;
    int k = l;
    while (i < n1 && j < n2) {
        if (L[i] <= R[j]) {
            arr[k] = L[i];
            i++;
        } else {
            arr[k] = R[j];
            j++;
        }
        k++;
    }

    while (i < n1) {
        arr[k] = L[i];
        i++;
        k++;
    }

    while (j < n2) {
        arr[k] = R[j];
        j++;
        k++;
    }

    delete[] L;
    delete[] R;
}

__device__ void merge_sort_kernel(int* arr, int l, int r) {
    if (l < r) {
        int m = l + (r - l) / 2;
        merge_sort_kernel(arr, l, m);
        merge_sort_kernel(arr, m + 1, r);
        merge_kernel(arr, l, m, r);
    }
}

__global__ void parallel_merge_sort_kernel(int* arr, int size) {
    merge_sort_kernel(arr, 0, size - 1);
}

// CUDA kernel for parallel Heap Sort
__device__ void max_heapify_kernel(int* arr, int n, int i) {
    int largest = i;
    int l = 2 * i + 1;
    int r = 2 * i + 2;

    if (l < n && arr[l] > arr[largest])
        largest = l;

    if (r < n && arr[r] > arr[largest])
        largest = r;

    if (largest != i) {
        int temp = arr[i];
        arr[i] = arr[largest];
        arr[largest] = temp;

        max_heapify_kernel(arr, n, largest);
    }
}

__global__ void parallel_heap_sort_kernel(int* arr, int size) {
    for (int i = size / 2 - 1; i >= 0; i--)
        max_heapify_kernel(arr, size, i);

    for (int i = size - 1; i >= 0; i--) {
        int temp = arr[0];
        arr[0] = arr[i];
        arr[i] = temp;

        max_heapify_kernel(arr, i, 0);
    }
}

// Serial Bubble Sort Execution
void serial_bubble_sort_execution(const vector<int>& arr) {
    vector<int> serial_arr(arr);
    auto start_serial = chrono::steady_clock::now();
    bubble_sort_serial(serial_arr);
    auto end_serial = chrono::steady_clock::now();
    chrono::duration<double> elapsed_serial = end_serial - start_serial;
    // cout << "Serial Bubble Sort Execution Time: " << elapsed_serial.count() * 1000 << " milliseconds" << endl;
}

// Serial Selection Sort Execution
void serial_selection_sort_execution(const vector<int>& arr) {
    vector<int> serial_arr(arr);
    auto start_serial = chrono::steady_clock::now();
    selection_sort_serial(serial_arr);
    auto end_serial = chrono::steady_clock::now();
    chrono::duration<double> elapsed_serial = end_serial - start_serial;
    // cout << "Serial Selection Sort Execution Time: " << elapsed_serial.count() * 1000 << " milliseconds" << endl;
}

// Serial Insertion Sort Execution
void serial_insertion_sort_execution(const vector<int>& arr) {
    vector<int> serial_arr(arr);
    auto start_serial = chrono::steady_clock::now();
    insertion_sort_serial(serial_arr);
    auto end_serial = chrono::steady_clock::now();
    chrono::duration<double> elapsed_serial = end_serial - start_serial;
    // cout << "Serial Insertion Sort Execution Time: " << elapsed_serial.count() * 1000 << " milliseconds" << endl;
}

// Serial Quick Sort Execution
void serial_quick_sort_execution(const vector<int>& arr) {
    vector<int> serial_arr(arr);
    auto start_serial = chrono::steady_clock::now();
    quick_sort_serial(serial_arr, 0, serial_arr.size() - 1);
    auto end_serial = chrono::steady_clock::now();
    chrono::duration<double> elapsed_serial = end_serial - start_serial;
    // cout << "Serial Quick Sort Execution Time: " << elapsed_serial.count() * 1000 << " milliseconds" << endl;
}

// Serial Merge Sort Execution
void serial_merge_sort_execution(const vector<int>& arr) {
    vector<int> serial_arr(arr);
    auto start_serial = chrono::steady_clock::now();
    merge_sort_serial(serial_arr, 0, serial_arr.size() - 1);
    auto end_serial = chrono::steady_clock::now();
    chrono::duration<double> elapsed_serial = end_serial - start_serial;
    // cout << "Serial Merge Sort Execution Time: " << elapsed_serial.count() * 1000 << " milliseconds" << endl;
}

// Serial Heap Sort Execution
void serial_heap_sort_execution(const vector<int>& arr) {
    vector<int> serial_arr(arr);
    auto start_serial = chrono::steady_clock::now();
    heap_sort_serial(serial_arr);
    auto end_serial = chrono::steady_clock::now();
    chrono::duration<double> elapsed_serial = end_serial - start_serial;
    // cout << "Serial Heap Sort Execution Time: " << elapsed_serial.count() * 1000 << " milliseconds" << endl;
}

// Parallel Bubble Sort Execution
void parallel_bubble_sort_execution(const vector<int>& arr) {
    int size = arr.size();
    int* d_arr;
    hipMalloc(&d_arr, size * sizeof(int));
    hipMemcpy(d_arr, arr.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    bubble_sort_kernel<<<1, size>>>(d_arr, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // cout << "Parallel Bubble Sort Execution Time: " << milliseconds << " milliseconds" << endl;
    hipFree(d_arr);
}

// Parallel Selection Sort Execution
void parallel_selection_sort_execution(const vector<int>& arr) {
    int size = arr.size();
    int* d_arr;
    hipMalloc(&d_arr, size * sizeof(int));
    hipMemcpy(d_arr, arr.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    selection_sort_kernel<<<1, size>>>(d_arr, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // cout << "Parallel Selection Sort Execution Time: " << milliseconds << " milliseconds" << endl;
    hipFree(d_arr);
}

// Parallel Insertion Sort Execution
void parallel_insertion_sort_execution(const vector<int>& arr) {
    int size = arr.size();
    int* d_arr;
    hipMalloc(&d_arr, size * sizeof(int));
    hipMemcpy(d_arr, arr.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    insertion_sort_kernel<<<1, size>>>(d_arr, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // cout << "Parallel Insertion Sort Execution Time: " << milliseconds << " milliseconds" << endl;
    hipFree(d_arr);
}

// Parallel Quick Sort Execution
void parallel_quick_sort_execution(const vector<int>& arr) {
    int size = arr.size();
    int* d_arr;
    hipMalloc(&d_arr, size * sizeof(int));
    hipMemcpy(d_arr, arr.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    parallel_quick_sort_kernel<<<1, size>>>(d_arr, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // cout << "Parallel Quick Sort Execution Time: " << milliseconds << " milliseconds" << endl;
    hipFree(d_arr);
}

// Parallel Merge Sort Execution
void parallel_merge_sort_execution(const vector<int>& arr) {
    int size = arr.size();
    int* d_arr;
    hipMalloc(&d_arr, size * sizeof(int));
    hipMemcpy(d_arr, arr.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    parallel_merge_sort_kernel<<<1, size>>>(d_arr, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // cout << "Parallel Merge Sort Execution Time: " << milliseconds << " milliseconds" << endl;
    hipFree(d_arr);
}

// Parallel Heap Sort Execution
void parallel_heap_sort_execution(const vector<int>& arr) {
    int size = arr.size();
    int* d_arr;
    hipMalloc(&d_arr, size * sizeof(int));
    hipMemcpy(d_arr, arr.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    parallel_heap_sort_kernel<<<1, size>>>(d_arr, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // cout << "Parallel Heap Sort Execution Time: " << milliseconds << " milliseconds" << endl;
    hipFree(d_arr);
}

int main() {
    int size;
    cout << "Enter the size of the array: ";
    cin >> size;

    vector<int> arr = generate_random_array(size);

    cout << "Algorithm\tSerial Time (ms)\tParallel Time (ms)\tSpeedup\t\tEfficiency" << endl;
    cout << "--------------------------------------------------------------------------------------------" << endl;

    // Bubble Sort
    auto start_serial_bubble = chrono::steady_clock::now();
    bubble_sort_serial(arr);
    auto end_serial_bubble = chrono::steady_clock::now();
    chrono::duration<double, milli> elapsed_serial_bubble = end_serial_bubble - start_serial_bubble;

    auto start_parallel_bubble = chrono::steady_clock::now();
    parallel_bubble_sort_execution(arr);
    auto end_parallel_bubble = chrono::steady_clock::now();
    chrono::duration<double, milli> elapsed_parallel_bubble = end_parallel_bubble - start_parallel_bubble;

    double speedup_bubble = elapsed_serial_bubble.count() / elapsed_parallel_bubble.count();
    double efficiency_bubble = speedup_bubble / 8; // Assuming 8 cores

    cout << "Bubble Sort\t" << fixed << setprecision(6) << setw(15) << elapsed_serial_bubble.count() << "\t   " << setw(15) << elapsed_parallel_bubble.count() << "\t\t" << setw(10) << speedup_bubble << "\t" << setw(10) << efficiency_bubble << endl;

    // Selection Sort
    auto start_serial_selection = chrono::steady_clock::now();
    selection_sort_serial(arr);
    auto end_serial_selection = chrono::steady_clock::now();
    chrono::duration<double, milli> elapsed_serial_selection = end_serial_selection - start_serial_selection;

    auto start_parallel_selection = chrono::steady_clock::now();
    parallel_selection_sort_execution(arr);
    auto end_parallel_selection = chrono::steady_clock::now();
    chrono::duration<double, milli> elapsed_parallel_selection = end_parallel_selection - start_parallel_selection;

    double speedup_selection = elapsed_serial_selection.count() / elapsed_parallel_selection.count();
    double efficiency_selection = speedup_selection / 8; // Assuming 8 cores

    cout << "Selection Sort\t" << fixed << setprecision(6) << setw(15) << elapsed_serial_selection.count() << "\t   " << setw(15) << elapsed_parallel_selection.count() << "\t\t" << setw(10) << speedup_selection << "\t" << setw(10) << efficiency_selection << endl;

    // Insertion Sort
    auto start_serial_insertion = chrono::steady_clock::now();
    insertion_sort_serial(arr);
    auto end_serial_insertion = chrono::steady_clock::now();
    chrono::duration<double, milli> elapsed_serial_insertion = end_serial_insertion - start_serial_insertion;

    auto start_parallel_insertion = chrono::steady_clock::now();
    parallel_insertion_sort_execution(arr);
    auto end_parallel_insertion = chrono::steady_clock::now();
    chrono::duration<double, milli> elapsed_parallel_insertion = end_parallel_insertion - start_parallel_insertion;

    double speedup_insertion = elapsed_serial_insertion.count() / elapsed_parallel_insertion.count();
    double efficiency_insertion = speedup_insertion / 8; // Assuming 8 cores

    cout << "Insertion Sort\t" << fixed << setprecision(6) << setw(15) << elapsed_serial_insertion.count() << "\t   " << setw(15) << elapsed_parallel_insertion.count() << "\t\t" << setw(10) << speedup_insertion << "\t" << setw(10) << efficiency_insertion << endl;

    // Quick Sort
    auto start_serial_quick = chrono::steady_clock::now();
    quick_sort_serial(arr, 0, arr.size() - 1);
    auto end_serial_quick = chrono::steady_clock::now();
    chrono::duration<double, milli> elapsed_serial_quick = end_serial_quick - start_serial_quick;

    auto start_parallel_quick = chrono::steady_clock::now();
    parallel_quick_sort_execution(arr);
    auto end_parallel_quick = chrono::steady_clock::now();
    chrono::duration<double, milli> elapsed_parallel_quick = end_parallel_quick - start_parallel_quick;

    double speedup_quick = elapsed_serial_quick.count() / elapsed_parallel_quick.count();
    double efficiency_quick = speedup_quick / 8; // Assuming 8 cores

    cout << "Quick Sort\t" << fixed << setprecision(6) << setw(15) << elapsed_serial_quick.count() << "\t   " << setw(15) << elapsed_parallel_quick.count() << "\t\t" << setw(10) << speedup_quick << "\t" << setw(10) << efficiency_quick << endl;

    // Merge Sort
    auto start_serial_merge = chrono::steady_clock::now();
    merge_sort_serial(arr, 0, arr.size() - 1);
    auto end_serial_merge = chrono::steady_clock::now();
    chrono::duration<double, milli> elapsed_serial_merge = end_serial_merge - start_serial_merge;

    auto start_parallel_merge = chrono::steady_clock::now();
    parallel_merge_sort_execution(arr);
    auto end_parallel_merge = chrono::steady_clock::now();
    chrono::duration<double, milli> elapsed_parallel_merge = end_parallel_merge - start_parallel_merge;

    double speedup_merge = elapsed_serial_merge.count() / elapsed_parallel_merge.count();
    double efficiency_merge = speedup_merge / 8; // Assuming 8 cores

    cout << "Merge Sort\t" << fixed << setprecision(6) << setw(15) << elapsed_serial_merge.count() << "\t   " << setw(15) << elapsed_parallel_merge.count() << "\t\t" << setw(10) << speedup_merge << "\t" << setw(10) << efficiency_merge << endl;

    // Heap Sort
    auto start_serial_heap = chrono::steady_clock::now();
    heap_sort_serial(arr);
    auto end_serial_heap = chrono::steady_clock::now();
    chrono::duration<double, milli> elapsed_serial_heap = end_serial_heap - start_serial_heap;

    auto start_parallel_heap = chrono::steady_clock::now();
    parallel_heap_sort_execution(arr);
    auto end_parallel_heap = chrono::steady_clock::now();
    chrono::duration<double, milli> elapsed_parallel_heap = end_parallel_heap - start_parallel_heap;

    double speedup_heap = elapsed_serial_heap.count() / elapsed_parallel_heap.count();
    double efficiency_heap = speedup_heap / 8; // Assuming 8 cores

    cout << "Heap Sort\t" << fixed << setprecision(6) << setw(15) << elapsed_serial_heap.count() << "\t   " << setw(15) << elapsed_parallel_heap.count() << "\t\t" << setw(10) << speedup_heap << "\t" << setw(10) << efficiency_heap << endl;

    return 0;
}
